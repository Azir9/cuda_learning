#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cassert>

void Matmul_imply(float *A_host, float *B_host, float *C_Host, int width, int blockSize);
extern "C" void matmulCuda(const float *A, const float *B, float *C,
                           int width, int blockSize)
{
    Matmul_imply(const_cast<float*>(A),
           const_cast<float*>(B),
           C,
           width,
           blockSize);
}

/*
       **
       **
       **    
        y
*****  **x

*/






__global__ void matmulkernel(float* A_device,float* B_device,float* C_device,int width){

    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    for (int k = 0; k < width; ++k)
        sum += A_device[x * width + k] * B_device[k * width + y];
        //这里索引为什么是这个
    C_device[x * width + y] = sum;
    
}

//summery 索引
/*A * B = C*/
void Matmul_imply(float *A_host ,float *B_host, float* C_Host,int width,int blockSize){
    /* set the size of matrix*/
    int size = width * width * sizeof(float);

    float *A_device;
    float *B_device;

    hipMalloc(&A_device,size);
    hipMalloc(&B_device,size);
    // memory copy
    hipMemcpy(A_device,A_host,size,hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(B_device,B_host,size,hipMemcpyKind::hipMemcpyHostToDevice);
    // result copy
    float *C_device;

    hipMalloc(&C_device,size); //create memory
    //kernel function

    dim3 blockDim(blockSize,blockSize);
    dim3 gridDim(width/blockSize,width/blockSize);
    matmulkernel <<<gridDim,blockDim>>> (A_device,B_device,C_device,width);

    //copy the result
    hipMemcpy(C_Host, C_device, size, hipMemcpyDeviceToHost);  

    //synchronize
    hipDeviceSynchronize();
    
    hipFree(C_device);
    hipFree(B_device);
    hipFree(A_device);
}
